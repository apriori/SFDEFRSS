/*
Copyright 2016 Fixstars Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

http ://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#include "sgm.hpp"
#include "census_transform.hpp"
#include "path_aggregation.hpp"
#include "winner_takes_all.hpp"

namespace sgm {

template <typename T, size_t MAX_DISPARITY>
class SemiGlobalMatching<T, MAX_DISPARITY>::Impl {

public:
	CensusTransform<T> m_census_left;
	CensusTransform<T> m_census_right;
	PathAggregation<MAX_DISPARITY> m_path_aggregation;
	WinnerTakesAll<MAX_DISPARITY> m_winner_takes_all;
public:
	Impl()
		: m_census_left()
		, m_census_right()
		, m_path_aggregation()
		, m_winner_takes_all()
	{ }

	void enqueue(
		output_type *dest_left,
		output_type *dest_right,
		const input_type *src_left,
		const input_type *src_right,
		int width,
		int height,
		int src_pitch,
		int dst_pitch,
		const StereoSGM::Parameters& param,
		hipStream_t stream)
	{
		m_census_left.enqueue(
			src_left, width, height, src_pitch, stream);
		m_census_right.enqueue(
			src_right, width, height, src_pitch, stream);
		m_path_aggregation.enqueue(
			m_census_left.get_output(),
			m_census_right.get_output(),
			width, height,
			param.path_type, param.P1, param.P2, param.min_disp,
			stream);
		m_winner_takes_all.enqueue(
			dest_left, dest_right,
			m_path_aggregation.get_output(),
			width, height, dst_pitch,
			param.uniqueness, param.subpixel, param.path_type,
			stream);
	}

	void enqueue2(
		output_type *dest_left,
		output_type *dest_right,
		const uint32_t * cost,
		int width,
		int height,
		const StereoSGM::Parameters& param,
		hipStream_t stream)
	{
		m_path_aggregation.enqueue2(
			cost,
			width, height,
			param.path_type, param.P1, param.P2, param.min_disp,
			stream);
		m_winner_takes_all.enqueue(
			dest_left, dest_right,
			m_path_aggregation.get_output(),
			width, height, width,
			param.uniqueness, param.subpixel, param.path_type,
			stream);
	}


	void enqueue3(
		uint8_t ** dest_left,
		const uint32_t * cost,
		int width,
		int height,
		const StereoSGM::Parameters& param,
		hipStream_t stream)
	{
		m_path_aggregation.enqueue2(
			cost,
			width, height,
			param.path_type, param.P1, param.P2, param.min_disp,
			stream);
		*dest_left = (uint8_t *)(m_path_aggregation.get_output());
	}


};


template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::SemiGlobalMatching()
	: m_impl(new Impl())
{ }

template <typename T, size_t MAX_DISPARITY>
SemiGlobalMatching<T, MAX_DISPARITY>::~SemiGlobalMatching() = default;


template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::execute(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		0);
	hipStreamSynchronize(0);
}

template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::enqueue(
	output_type *dest_left,
	output_type *dest_right,
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch,
	int dst_pitch,
	const StereoSGM::Parameters& param,
	hipStream_t stream)
{
	m_impl->enqueue(
		dest_left, dest_right,
		src_left, src_right,
		width, height,
		src_pitch, dst_pitch,
		param,
		stream);
}

template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::FeatureTransform(
	const input_type *src_left,
	const input_type *src_right,
	int width,
	int height,
	int src_pitch)
{
	m_impl->m_census_left.enqueue(
		src_left, width, height, src_pitch, 0);
	m_impl->m_census_right.enqueue(
		src_right, width, height, src_pitch, 0);
	hipStreamSynchronize(0);
}

template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::InvolveSmooth(
	output_type *dest_left,
	output_type *dest_right,
	const uint32_t * cost_volume, 
	int width, 
	int height, 
	StereoSGM::Parameters& param)
{

	m_impl->enqueue2(
		dest_left, dest_right,
		cost_volume,
		width, height,
		param,
		0);
	hipStreamSynchronize(0);

}


template <typename T, size_t MAX_DISPARITY>
void SemiGlobalMatching<T, MAX_DISPARITY>::InvolveSmooth2(
	uint8_t ** finalcost,
	const uint32_t * cost_volume, 
	int width, 
	int height, 
	StereoSGM::Parameters& param)
{

	m_impl->enqueue3(
		finalcost,
		cost_volume,
		width, height,
		param,
		0);
	hipStreamSynchronize(0);
}


template <typename T, size_t MAX_DISPARITY>
const feature_type * SemiGlobalMatching<T, MAX_DISPARITY>::get_left_output() const {
	return m_impl->m_census_left.get_output();
}

template <typename T, size_t MAX_DISPARITY>
const feature_type * SemiGlobalMatching<T, MAX_DISPARITY>::get_right_output() const {
	return m_impl->m_census_right.get_output();
}

template class SemiGlobalMatching<uint8_t,   64>;
template class SemiGlobalMatching<uint8_t,  128>;
template class SemiGlobalMatching<uint8_t,  256>;
template class SemiGlobalMatching<uint16_t,  64>;
template class SemiGlobalMatching<uint16_t, 128>;
template class SemiGlobalMatching<uint16_t, 256>;

}
